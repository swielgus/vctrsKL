#include "hip/hip_runtime.h"

#include "ImageData.hpp"
#include "ImageComponentLabeling.hpp"

__global__ void
convertToYUV(ImageData::color_type* inputR, ImageData::color_type* inputG, ImageData::color_type* inputB,
             ImageData::color_type* outputY, ImageData::color_type* outputU, ImageData::color_type* outputV,
             const std::size_t width, const std::size_t height)
{
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j = threadIdx.y + (blockIdx.y * blockDim.y);
    if(i < height && j < width)
    {
        std::size_t idx = j + i * width;

        ImageData::color_type red = inputR[idx];
        ImageData::color_type green = inputG[idx];
        ImageData::color_type blue = inputB[idx];

        outputY[idx] = static_cast<ImageData::color_type>(0.299 * red + 0.587 * green + 0.114 * blue);
        outputU[idx] = static_cast<ImageData::color_type>((-0.169 * red - 0.331 * green + 0.5 * blue) + 128);
        outputV[idx] = static_cast<ImageData::color_type>((0.5 * red - 0.419 * green - 0.081 * blue) + 128);
    }
}

const ImageData::color_type* ImageData::getGPUAddressOfYColorData() const
{
    return d_colorYData;
}
const ImageData::color_type* ImageData::getGPUAddressOfUColorData() const
{
    return d_colorUData;
}
const ImageData::color_type* ImageData::getGPUAddressOfVColorData() const
{
    return d_colorVData;
}
const int* ImageData::getGPUAddressOfLabelData() const
{
    return d_componentLabels;
}

std::size_t ImageData::getWidth() const
{
    return internalImage.get_width();
}

std::size_t ImageData::getHeight() const
{
    return internalImage.get_height();
}

void ImageData::loadImage(std::string filename)
{
    internalImage.read(filename);
    allocatePixelDataOnDevice();
    createLabelsForSimilarPixels();
}

const png::rgb_pixel& ImageData::getPixel(std::size_t x, std::size_t y) const
{
    return internalImage.get_row(x).at(y);
}

const ImageData::color_type& ImageData::getPixelRed(std::size_t x, std::size_t y) const
{
    return getPixel(x, y).red;
}

const ImageData::color_type& ImageData::getPixelGreen(std::size_t x, std::size_t y) const
{
    return getPixel(x, y).green;
}

const ImageData::color_type& ImageData::getPixelBlue(std::size_t x, std::size_t y) const
{
    return getPixel(x, y).blue;
}

ImageData::color_type ImageData::getPixelY(std::size_t x, std::size_t y) const
{
    color_type* pixelYValue = new color_type;
    hipMemcpy( pixelYValue, d_colorYData+(y+x*this->getWidth()), sizeof(color_type), hipMemcpyDeviceToHost);

    color_type result = *pixelYValue;
    delete pixelYValue;
    return result;
}

ImageData::color_type ImageData::getPixelU(std::size_t x, std::size_t y) const
{
    color_type* pixelUValue = new color_type;
    hipMemcpy( pixelUValue, d_colorUData+(y+x*this->getWidth()), sizeof(color_type), hipMemcpyDeviceToHost);

    color_type result = *pixelUValue;
    delete pixelUValue;
    return result;
}

ImageData::color_type ImageData::getPixelV(std::size_t x, std::size_t y) const
{
    color_type* pixelVValue = new color_type;
    hipMemcpy( pixelVValue, d_colorVData+(y+x*this->getWidth()), sizeof(color_type), hipMemcpyDeviceToHost);

    color_type result = *pixelVValue;
    delete pixelVValue;
    return result;
}

ImageData::ImageData(std::string filename)
        : internalImage{}, d_colorYData{nullptr}, d_colorUData{nullptr}, d_colorVData{nullptr},
          d_componentLabels{nullptr}
{
    this->loadImage(filename);
}

ImageData::ImageData()
        : internalImage{}, d_colorYData{nullptr}, d_colorUData{nullptr}, d_colorVData{nullptr},
          d_componentLabels{nullptr}

{}

ImageData::~ImageData()
{
    freeDeviceData();
    //hipDeviceReset();
}

void ImageData::freeDeviceData()
{
    hipFree(d_colorYData);
    hipFree(d_colorUData);
    hipFree(d_colorVData);
    hipFree(d_componentLabels);
}

std::vector< std::vector<int> > ImageData::getLabelValues() const
{
    const std::size_t width = getWidth();
    const std::size_t height = getHeight();

    std::vector< std::vector<int> > result;
    result.resize(height);
    for(std::vector<int>& row : result)
        row.resize(width);

    int* pixelLabels = new int[width * height];
    hipMemcpy(pixelLabels, d_componentLabels, width * height * sizeof(int), hipMemcpyDeviceToHost);

    for(std::size_t x = 0; x < height; ++x)
    for(std::size_t y = 0; y < width; ++y)
    {
        result[x][y] = *(pixelLabels + (y + x * width));
    }

    delete[] pixelLabels;

    return result;
}

void ImageData::allocatePixelDataOnDevice()
{
    //hipEvent_t start, stop;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);

    freeDeviceData();
    
    std::size_t height = this->getHeight();
    std::size_t width = this->getWidth();

    color_type* d_inputRData = nullptr;
    color_type* d_inputGData = nullptr;
    color_type* d_inputBData = nullptr;

    hipMalloc( &d_inputRData, width * height * sizeof(color_type));
    hipMalloc( &d_inputGData, width * height * sizeof(color_type));
    hipMalloc( &d_inputBData, width * height * sizeof(color_type));

    int k = 0;
    for(std::size_t i = 0; i < height; ++i)
    for(std::size_t j = 0; j < width; ++j)
    {
        const color_type& currentPixelR = this->getPixelRed(i,j);
        const color_type& currentPixelG = this->getPixelGreen(i,j);
        const color_type& currentPixelB = this->getPixelBlue(i,j);
        hipMemcpy( d_inputRData+k, &currentPixelR, sizeof(color_type), hipMemcpyHostToDevice );
        hipMemcpy( d_inputGData+k, &currentPixelG, sizeof(color_type), hipMemcpyHostToDevice );
        hipMemcpy( d_inputBData+k, &currentPixelB, sizeof(color_type), hipMemcpyHostToDevice );
        ++k;
    }

    hipMalloc( &d_colorYData, width * height * sizeof(color_type));
    hipMalloc( &d_colorUData, width * height * sizeof(color_type));
    hipMalloc( &d_colorVData, width * height * sizeof(color_type));



    dim3 dimBlock(16, 16);
    dim3 dimGrid((height + dimBlock.x -1)/dimBlock.x,
                 (width + dimBlock.y -1)/dimBlock.y);

    convertToYUV<<<dimGrid, dimBlock>>>(d_inputRData, d_inputGData, d_inputBData,
                                        d_colorYData, d_colorUData, d_colorVData, width, height);
    hipDeviceSynchronize();

    hipFree(d_inputRData);
    hipFree(d_inputGData);
    hipFree(d_inputBData);
}

void ImageData::createLabelsForSimilarPixels()
{
    std::size_t height = this->getHeight();
    std::size_t width = this->getWidth();
    hipMalloc( &d_componentLabels, width * height * sizeof(int));

    int blockSide = 16;
    dim3 dimBlock(blockSide, blockSide);
    dim3 dimGrid((height + dimBlock.y - 1)/dimBlock.y, (width + dimBlock.x - 1)/dimBlock.x);

    int numberOfPixelsPerBlock = dimBlock.x * dimBlock.y;

    //solve components in local squares
    ImageComponentLabeling::createLocalComponentLabels <<<dimGrid, dimBlock, (numberOfPixelsPerBlock * sizeof(int))+(3 * numberOfPixelsPerBlock * sizeof(Color::byte))>>>(
        d_colorYData, d_colorUData, d_colorVData, d_componentLabels, width, height);
    hipDeviceSynchronize();

    //merge small results into bigger groups
    while( (blockSide < width || blockSide < height) )
    {
        //compute the number of tiles that are going to be merged in a single thread block
        int numberOfTileRows = 4;
        int numberOfTileCols = 4;

        if(numberOfTileCols * blockSide > width)
            numberOfTileCols = (width + blockSide - 1) / blockSide;

        if(numberOfTileRows * blockSide > height)
            numberOfTileRows = (height + blockSide - 1) / blockSide;

        int threadsPerTile = 32;
        if(blockSide < threadsPerTile)
            threadsPerTile = blockSide;

        dim3 block(numberOfTileRows, numberOfTileCols, threadsPerTile);
        dim3 grid((height + (numberOfTileRows * blockSide) - 1) / (numberOfTileRows * blockSide),
                  (width + (numberOfTileCols * blockSide) - 1) / (numberOfTileCols * blockSide));
        ImageComponentLabeling::mergeSolutionsOnBlockBorders<<<grid, block>>>(
                d_colorYData, d_colorUData, d_colorVData, d_componentLabels, width, height, blockSide);

        if(numberOfTileCols > numberOfTileRows)
            blockSide = numberOfTileCols * blockSide;
        else
            blockSide = numberOfTileRows * blockSide;

        //TODO update labels on borders
    }

    //update all labels
    ImageComponentLabeling::flattenAllEquivalenceTrees<<<dimGrid, dimBlock>>>(d_componentLabels, width, height);
}
