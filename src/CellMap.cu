#include "CellMap.hpp"
#include "CellMapConstructing.hpp"

CellMap::CellMap(const PixelGraph& graph)
    : sourceGraph{graph}, d_cellData{nullptr}
{
    constructPixelCells();
}

CellMap::~CellMap()
{
    freeDeviceData();
}

void CellMap::freeDeviceData()
{
    hipFree(d_cellData);
}

std::vector< std::vector<CellSideType> > CellMap::getCellTypes() const
{
    const std::size_t width = sourceGraph.getWidth() + 1;
    const std::size_t height = sourceGraph.getHeight() + 1;

    std::vector< std::vector<CellSideType> > result;
    result.resize(height);
    for(std::vector<cell_type>& row : result)
        row.resize(width);

    CellSide* cellSideValues = new CellSide[width * height];
    hipMemcpy(cellSideValues, d_cellData, width * height * sizeof(CellSide), hipMemcpyDeviceToHost);

    Cell::byte firstThreeBitsMask = 7;
    for(std::size_t x = 0; x < height; ++x)
        for(std::size_t y = 0; y < width; ++y)
        {
            result[x][y] = static_cast<CellSideType>(((cellSideValues + (y + x * width))->type) & firstThreeBitsMask);
        }

    delete[] cellSideValues;

    return result;
}

void CellMap::constructPixelCells()
{
    const std::size_t width = sourceGraph.getWidth() + 1;
    const std::size_t height = sourceGraph.getHeight() + 1;
    hipMalloc( &d_cellData, width * height * sizeof(CellSide));

    const PixelGraph::edge_type* d_graphData = sourceGraph.getGPUAddressOfGraphData();

    dim3 dimBlock(16, 16);
    dim3 dimGrid((height + dimBlock.x -1)/dimBlock.x,
                 (width + dimBlock.y -1)/dimBlock.y);
    CellMapConstructing::createCells<<<dimGrid, dimBlock>>>(d_cellData, d_graphData, width, height);
    hipDeviceSynchronize();
}