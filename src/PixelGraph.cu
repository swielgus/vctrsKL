#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "PixelGraph.hpp"
#include "CrossingResolving.h"

__device__ PixelGraph::color_type getColorDifference(const PixelGraph::color_type& a, const PixelGraph::color_type& b)
{
    return (a < b) ? (b-a) : (a-b);
}

__device__ bool areYUVColorsSimilar(const PixelGraph::color_type& aY, const PixelGraph::color_type& aU,
                                  const PixelGraph::color_type& aV, const PixelGraph::color_type& bY,
                                  const PixelGraph::color_type& bU, const PixelGraph::color_type& bV)
{
    const PixelGraph::color_type thresholdY = 48;
    const PixelGraph::color_type thresholdU = 7;
    const PixelGraph::color_type thresholdV = 6;

    //return (getColorDifference(aY, bY) <= thresholdY) && (getColorDifference(aU, bU) <= thresholdU) &&
    //       (getColorDifference(aV, bV) <= thresholdV);
    return abs(aY - bY) <= thresholdY && abs(aU - bU) <= thresholdU && abs(aV - bV) <= thresholdV;
}

__global__ void
createConnections(PixelGraph::color_type* edges, const PixelGraph::color_type* colorY,
                  const PixelGraph::color_type* colorU, const PixelGraph::color_type* colorV,
                  const std::size_t* dim, const PixelGraph::color_type* directions)
{
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j = threadIdx.y + (blockIdx.y * blockDim.y);
    if(i < dim[0] && j < dim[1])
    {
        std::size_t idx = j + i * dim[1];
        edges[idx] = 0;

        for(int iMod = -1; iMod <= 1; ++iMod)
        for(int jMod = -1; jMod <= 1; ++jMod)
        {
            int iNew = i+iMod;
            int jNew = j+jMod;

            if( (iNew != i || jNew != j) && (iNew >= 0 && iNew < dim[0]) && (jNew >= 0 && jNew < dim[1]) )
            {
                std::size_t comparedIdx = iNew * dim[1] + jNew;
                if(areYUVColorsSimilar(colorY[idx],colorU[idx],colorV[idx],
                                       colorY[comparedIdx],colorU[comparedIdx],colorV[comparedIdx]))
                {
                    /* graph directions relative to point x:
                     *  1 | 128 | 64
                     * --------------
                     *  2 |  x  | 32
                     * --------------
                     *  4 |  8  | 16
                     */
                    edges[idx] += directions[(iMod+1)*3 + (jMod+1)];
                }
            }
        }
    }
}

__global__ void resolveCrossings(PixelGraph::color_type* edges, const std::size_t* dim,
                                 const PixelGraph::color_type* directions)
{
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j = threadIdx.y + (blockIdx.y * blockDim.y);
    if(i < dim[0]-1 && j < dim[1]-1)
    {
        std::size_t idx = j + i * dim[1];

        PixelGraph::color_type upperLeftConnected[12] = {255,48,24,255,56,255,255,255,255,40,40,255};
        PixelGraph::color_type lowerLeftConnected[12] = {96,255,255,192,255,255,224,255,255,160,255,160};
        PixelGraph::color_type upperRightConnected[12] = {6,255,255,12,255,14,255,255,10,255,10,255};
        PixelGraph::color_type lowerRightConnected[12] = {255,3,129,255,255,255,255,131,130,255,255,130};

        int k = 0;
        bool squareIsNotConnected = true;
        while(k < 12 && squareIsNotConnected)
        {
            squareIsNotConnected = !(
                    ((upperLeftConnected[k] == 255) || ((upperLeftConnected[k] & edges[idx]) == upperLeftConnected[k]))
                    && ((upperRightConnected[k] == 255) ||
                        ((upperRightConnected[k] & edges[idx + 1]) == upperRightConnected[k]))
                    && ((lowerLeftConnected[k] == 255) ||
                        ((lowerLeftConnected[k] & edges[idx + dim[1]]) == lowerLeftConnected[k]))
                    && ((lowerRightConnected[k] == 255) ||
                        ((lowerRightConnected[k] & edges[idx + dim[1] + 1]) == lowerRightConnected[k]))
            );
            ++k;
        }

        if(!squareIsNotConnected)
        {
            CrossingResolving::doAtomicOr(&edges[idx], directions[5] + directions[7]);
            CrossingResolving::doAtomicOr(&edges[idx+1], directions[3] + directions[7]);
            CrossingResolving::doAtomicOr(&edges[idx+dim[1]], directions[1] + directions[5]);
            CrossingResolving::doAtomicOr(&edges[idx+dim[1]+1], directions[1] + directions[3]);

            CrossingResolving::doAtomicAnd(&edges[idx], ~directions[8]);
            CrossingResolving::doAtomicAnd(&edges[idx+dim[1]+1], ~directions[0]);
            CrossingResolving::doAtomicAnd(&edges[idx+1], ~directions[6]);
            CrossingResolving::doAtomicAnd(&edges[idx+dim[1]], ~directions[2]);
        }



    }
}

PixelGraph::PixelGraph(const ImageData& image)
    : sourceImage{image}, d_pixelConnections{nullptr}, d_pixelDirections{nullptr}
{
    constructGraph();
}

PixelGraph::~PixelGraph()
{
    freeDeviceData();
    //hipDeviceReset();
}

void PixelGraph::freeDeviceData()
{
    hipFree(d_pixelConnections);
    hipFree(d_pixelDirections);
}

void PixelGraph::constructGraph()
{
    //hipEvent_t start, stop;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);

    freeDeviceData();

    const PixelGraph::color_type directions[9] = {1,128,64,2,0,32,4,8,16};
    hipMalloc( &d_pixelDirections, 9 * sizeof(color_type));
    hipMemcpy( d_pixelDirections, &directions, 9 * sizeof(color_type), hipMemcpyHostToDevice );

    const std::size_t width = sourceImage.getWidth();
    const std::size_t height = sourceImage.getHeight();
    hipMalloc( &d_pixelConnections, width * height * sizeof(color_type));

    dim3 dimBlock(16, 16);
    dim3 dimGrid((height + dimBlock.x -1)/dimBlock.x,
                 (width + dimBlock.y -1)/dimBlock.y);

    //hipEventRecord(start);
    createConnections<<<dimGrid, dimBlock>>>(d_pixelConnections, sourceImage.getGPUAddressOfYColorData(),
                                    sourceImage.getGPUAddressOfUColorData(), sourceImage.getGPUAddressOfVColorData(),
                                    sourceImage.getGPUAddressOfDimensionsData(), d_pixelDirections);
    hipDeviceSynchronize();
    //hipEventRecord(stop);

    //hipEventSynchronize(stop);
    //float milliseconds = 0;
    //hipEventElapsedTime(&milliseconds, start, stop);
    //printf("time:%f\n", milliseconds);
}

std::vector< std::vector<PixelGraph::color_type> > PixelGraph::getEdgeValues() const
{
    const std::size_t width = sourceImage.getWidth();
    const std::size_t height = sourceImage.getHeight();

    std::vector<std::vector<PixelGraph::color_type>> result;
    result.resize(height);
    for(std::vector<PixelGraph::color_type>& row : result)
        row.resize(width);

    color_type* pixelDirection = new color_type[width * height];
    hipMemcpy(pixelDirection, d_pixelConnections, width * height * sizeof(color_type), hipMemcpyDeviceToHost);

    for(std::size_t x = 0; x < height; ++x)
    for(std::size_t y = 0; y < width; ++y)
    {
        result[x][y] = *(pixelDirection + (y + x * width));
    }

    delete[] pixelDirection;

    return result;
}

void PixelGraph::resolveUnnecessaryDiagonals()
{
    const std::size_t width = sourceImage.getWidth();
    const std::size_t height = sourceImage.getHeight();
    dim3 dimBlock(16, 16);
    dim3 dimGrid((height + dimBlock.x -1)/dimBlock.x,
                 (width + dimBlock.y -1)/dimBlock.y);
    resolveCrossings<<<dimGrid, dimBlock>>>(d_pixelConnections, sourceImage.getGPUAddressOfDimensionsData(),
                                            d_pixelDirections);
    hipDeviceSynchronize();
}