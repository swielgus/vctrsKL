#include "PixelGraph.hpp"
#include "GraphCrossResolving.hpp"
#include "GraphConstructing.hpp"

PixelGraph::PixelGraph(const ImageData& image)
    : sourceImage{image}, d_pixelConnections{nullptr}, d_graphInfo{nullptr}
{
    constructGraph();
}

PixelGraph::~PixelGraph()
{
    freeDeviceData();
    //hipDeviceReset();
}

void PixelGraph::freeDeviceData()
{
    hipFree(d_pixelConnections);
    hipFree(d_graphInfo);
}

void PixelGraph::constructGraph()
{
    freeDeviceData();

    const std::size_t width = sourceImage.getWidth();
    const std::size_t height = sourceImage.getHeight();
    const int* addressOfLabelData = sourceImage.getGPUAddressOfLabelData();
    hipMalloc( &d_pixelConnections, width * height * sizeof(edge_type));

    //PixelGraphInfo graphInfo{d_pixelConnections, width, height};
    //hipMalloc( &d_graphInfo, sizeof(PixelGraphInfo));
    //hipMemcpy(d_graphInfo, &graphInfo, sizeof(PixelGraphInfo), hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((height + dimBlock.x -1)/dimBlock.x,
                 (width + dimBlock.y -1)/dimBlock.y);
    GraphConstructing::createConnections<<<dimGrid, dimBlock>>>(d_pixelConnections, sourceImage.getGPUAddressOfLabelData(), width, height);
    hipDeviceSynchronize();
}

std::vector< std::vector<PixelGraph::edge_type> > PixelGraph::getEdgeValues() const
{
    const std::size_t width = sourceImage.getWidth();
    const std::size_t height = sourceImage.getHeight();

    std::vector< std::vector<edge_type> > result;
    result.resize(height);
    for(std::vector<edge_type>& row : result)
        row.resize(width);

    edge_type* pixelDirection = new edge_type[width * height];
    hipMemcpy(pixelDirection, d_pixelConnections, width * height * sizeof(edge_type), hipMemcpyDeviceToHost);

    for(std::size_t x = 0; x < height; ++x)
    for(std::size_t y = 0; y < width; ++y)
    {
        result[x][y] = *(pixelDirection + (y + x * width));
    }

    delete[] pixelDirection;

    return result;
}

void PixelGraph::resolveCrossings()
{
    const std::size_t width = sourceImage.getWidth();
    const std::size_t height = sourceImage.getHeight();
    dim3 dimBlock(32, 32);
    dim3 dimGrid((height + dimBlock.x -1)/dimBlock.x,
                 (width + dimBlock.y -1)/dimBlock.y);

    GraphCrossResolving::resolveCriticalCrossings<<<dimGrid, dimBlock>>>(d_pixelConnections, sourceImage.getGPUAddressOfLabelData(), width, height);
    hipDeviceSynchronize();
}